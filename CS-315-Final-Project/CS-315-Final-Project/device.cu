#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "RGB.h"
#include <math.h>
#include <stdio.h>
#include <iostream>

/**
* Helper function to calculate the greyscale value based on R, G, and B
*/
__device__ int greyscale(BYTE red, BYTE green, BYTE blue)
{
	int grey = 0.3 * red + 0.59 * green + 0 * 11 * blue; // calculate grey scale
	return min(grey, 255);
}

/**
* Kernel for executing on GPY
*/
__global__ void greyscaleKernel(RGB* d_pixels, int height, int width)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x; // width
	int y = blockIdx.y * blockDim.y + threadIdx.y; // height

	if (y >= height || y >= width)
		return;

	int index = y * width + x;

	int grey = greyscale(d_pixels[index].red, d_pixels[index].green, d_pixels[index].blue); // calculate grey scale

	d_pixels[index].red = grey;
	d_pixels[index].green = grey;
	d_pixels[index].blue = grey;

}

// Kernel to blur an image on the GPU
__global__ void blurKernel(RGB* d_pixels, int height, int width)
{
	// determine the current pixel
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	// if the pixel is still inside the dimensions of the image
	if (col < width && row < height) {
		// temp values to hold the new rgb values for our blurred pixel
		int redVal = 0;
		int greenVal = 0;
		int blueVal = 0;
		// a count of how many pixels were used to determine the blurred values
		int pixels = 0;

		// change the blurRow and blurCol for a different stencil to make the image more or less blurred
		for (int blurRow = -5; blurRow <= 5; ++blurRow) {
			for (int blurCol = -5; blurCol <= 5; ++blurCol) {
				int curRow = row + blurRow;
				int curCol = col + blurCol;
				// check to make sure that this is an existing pixel that we want to use for blurring
				if (curRow > -1 && curRow < height && curCol > -1 && curCol < width) {
					// add the rgb values of neighboring pixels to the temp values
					redVal += d_pixels[curRow * width + curCol].red;
					greenVal += d_pixels[curRow * width + curCol].green;
					blueVal += d_pixels[curRow * width + curCol].blue;
					// increment pixels to show how many pixels we looked at
					pixels++;
				}
			}
		}
		// average the rgb values by dividing by the count of pixels
		d_pixels[row * width + col].red = (unsigned char)(redVal / pixels);
		d_pixels[row * width + col].green = (unsigned char)(greenVal / pixels);
		d_pixels[row * width + col].blue = (unsigned char)(blueVal / pixels);
	}
}

// Sobel Filter
// |Gx(x,y)| = -P(x-1,y-1) + -2 *P(x-1,y) + -P(x-1,y+1) + P(x+1,y-1) + 2 * P(x + 1, y) + P(x + 1, y + 1)
// |Gy(x,y)| = P(x-1,y-1) + 2*P(x,y-1) + P(x+1,y-1) + -P(x-1,y+1) + �2 * P(x, y + 1) - P(x + 1, y + 1)
__global__ void edgeDetectionKernel(RGB* d_pixels, RGB* d_result, int height, int width)
{
	// determine the current pixel
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	float reddx, reddy;
	float greendx, greendy;
	float bluedx, bluedy;

	if (col > 0 && row > 0 && col < width - 1 && row < height - 1) {
		//red
		reddx = (-1 * d_pixels[(row - 1) * width + (col - 1)].red) + (-2 * d_pixels[row * width + (col - 1)].red) + (-1 * d_pixels[(row + 1) * width + (col - 1)].red) + (d_pixels[(row - 1) * width + (col + 1)].red) + (2 * d_pixels[row * width + (col + 1)].red) + (d_pixels[(row + 1) * width + (col + 1)].red);
		reddy = (d_pixels[(row - 1) * width + (col - 1)].red) + (2 * d_pixels[(row - 1) * width + col].red) + (d_pixels[(row - 1) * width + (col + 1)].red) + (-1 * d_pixels[(row + 1) * width + (col - 1)].red) + (-2 * d_pixels[(row + 1) * width + col].red) + (-1 * d_pixels[(row + 1) * width + (col + 1)].red);

		d_result[row * width + col].red = (unsigned char)(sqrt((reddx * reddx) + (reddy * reddy)));

		//green
		greendx = (-1 * d_pixels[(row - 1) * width + (col - 1)].green) + (-2 * d_pixels[row * width + (col - 1)].green) + (-1 * d_pixels[(row + 1) * width + (col - 1)].green) + (d_pixels[(row - 1) * width + (col + 1)].green) + (2 * d_pixels[row * width + (col + 1)].green) + (d_pixels[(row + 1) * width + (col + 1)].green);
		greendy = (d_pixels[(row - 1) * width + (col - 1)].green) + (2 * d_pixels[(row - 1) * width + col].green) + (d_pixels[(row - 1) * width + (col + 1)].green) + (-1 * d_pixels[(row + 1) * width + (col - 1)].green) + (-2 * d_pixels[(row + 1) * width + col].green) + (-1 * d_pixels[(row + 1) * width + (col + 1)].green);

		d_result[row * width + col].green = (unsigned char)(sqrt((greendx * greendx) + (greendy * greendy)));

		//blue
		bluedx = (-1 * d_pixels[(row - 1) * width + (col - 1)].blue) + (-2 * d_pixels[row * width + (col - 1)].blue) + (-1 * d_pixels[(row + 1) * width + (col - 1)].blue) + (d_pixels[(row - 1) * width + (col + 1)].blue) + (2 * d_pixels[row * width + (col + 1)].blue) + (d_pixels[(row + 1) * width + (col + 1)].blue);
		bluedy = (d_pixels[(row - 1) * width + (col - 1)].blue) + (2 * d_pixels[(row - 1) * width + col].blue) + (d_pixels[(row - 1) * width + (col + 1)].blue) + (-1 * d_pixels[(row + 1) * width + (col - 1)].blue) + (-2 * d_pixels[(row + 1) * width + col].blue) + (-1 * d_pixels[(row + 1) * width + (col + 1)].blue);

		d_result[row * width + col].blue = (unsigned char)(sqrt((bluedx * bluedx) + (bluedy * bluedy)));

	}
}

// -1  -1  -1
// -1   8  -1
// -1  -1  -1
// Laplacian filter
__global__ void laplacianKernel(RGB* d_pixels, RGB* d_result, int height, int width)
{
	// determine the current pixel
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	float red, green, blue;

	// apply the filter shown above to red, green, and blue for the current pixel
	if (col > 0 && row > 0 && col < width - 1 && row < height - 1) {
		red = (-1 * d_pixels[(row - 1) * width + (col - 1)].red) + (-1 * d_pixels[(row - 1) * width + col].red) + (-1 * d_pixels[(row - 1) * width + (col + 1)].red) + (-1 * d_pixels[row * width + (col - 1)].red) + (8 * d_pixels[row * width + col].red) + (-1 * d_pixels[row * width + (col + 1)].red) + (-1 * d_pixels[(row + 1) * width + (col - 1)].red) + (-1 * d_pixels[(row + 1) * width + col].red) + (-1 * d_pixels[(row + 1) * width + (col - 1)].red);
		green = (-1 * d_pixels[(row - 1) * width + (col - 1)].green) + (-1 * d_pixels[(row - 1) * width + col].green) + (-1 * d_pixels[(row - 1) * width + (col + 1)].green) + (-1 * d_pixels[row * width + (col - 1)].green) + (8 * d_pixels[row * width + col].green) + (-1 * d_pixels[row * width + (col + 1)].green) + (-1 * d_pixels[(row + 1) * width + (col - 1)].green) + (-1 * d_pixels[(row + 1) * width + col].green) + (-1 * d_pixels[(row + 1) * width + (col - 1)].green);
		blue = (-1 * d_pixels[(row - 1) * width + (col - 1)].blue) + (-1 * d_pixels[(row - 1) * width + col].blue) + (-1 * d_pixels[(row - 1) * width + (col + 1)].blue) + (-1 * d_pixels[row * width + (col - 1)].blue) + (8 * d_pixels[row * width + col].blue) + (-1 * d_pixels[row * width + (col + 1)].blue) + (-1 * d_pixels[(row + 1) * width + (col - 1)].blue) + (-1 * d_pixels[(row + 1) * width + col].blue) + (-1 * d_pixels[(row + 1) * width + (col - 1)].blue);

		d_result[row * width + col].red = red;
		d_result[row * width + col].green = green;
		d_result[row * width + col].blue = blue;
	}
}

// increase the red, green, or blue contrast without going out of bounds (0-255)
__global__ void contrastKernel(RGB* d_pixels, int height, int width, int rincrease, int gincrease, int bincrease)
{
	// determine the current pixel
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	if (col < width && row < height) {
		int index = row * width + col;
		//red
		if (d_pixels[index].red + rincrease < 256 && d_pixels[index].red + rincrease > -1) {
			d_pixels[index].red += rincrease;
		}
		else if (d_pixels[index].red + rincrease > 255) {
			d_pixels[index].red = 255;
		}
		else if (d_pixels[index].red + rincrease < 0) {
			d_pixels[index].red = 0;
		}
		//green
		if (d_pixels[index].green + gincrease < 256 && d_pixels[index].green + gincrease > -1) {
			d_pixels[index].green += gincrease;
		}
		else if (d_pixels[index].green + gincrease > 255) {
			d_pixels[index].green = 255;
		}
		else if (d_pixels[index].green + gincrease < 0) {
			d_pixels[index].green = 0;
		}
		// blue
		if (d_pixels[index].blue + bincrease < 256 && d_pixels[index].blue + bincrease > -1) {
			d_pixels[index].blue += bincrease;
		}
		else if (d_pixels[index].blue + bincrease > 255) {
			d_pixels[index].blue = 255;
		}
		else if (d_pixels[index].blue + bincrease < 0) {
			d_pixels[index].blue = 0;
		}
	}
}

// increase the brightness of an image without going out of bounds (0-255)
__global__ void brightnessKernel(RGB* d_pixels, int height, int width, int bright)
{
	// determine the current pixel
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	if (col < width && row < height) {
		int index = row * width + col;
		//red
		if (d_pixels[index].red * bright < 256 && d_pixels[index].red * bright > -1) {
			d_pixels[index].red *= bright;
		}
		else if (d_pixels[index].red * bright > 255) {
			d_pixels[index].red = 255;
		}
		else if (d_pixels[index].red * bright < 0) {
			d_pixels[index].red = 0;
		}
		//green
		if (d_pixels[index].green * bright < 256 && d_pixels[index].green * bright > -1) {
			d_pixels[index].green *= bright;
		}
		else if (d_pixels[index].green * bright > 255) {
			d_pixels[index].green = 255;
		}
		else if (d_pixels[index].green * bright < 0) {
			d_pixels[index].green = 0;
		}
		//blue
		if (d_pixels[index].blue * bright < 256 && d_pixels[index].blue * bright > -1) {
			d_pixels[index].blue *= bright;
		}
		else if (d_pixels[index].blue * bright > 255) {
			d_pixels[index].blue = 255;
		}
		else if (d_pixels[index].blue * bright < 0) {
			d_pixels[index].blue = 0;
		}
	}
}

//finds the gradient and edge direction of the image (used for canny)
__global__ void GradiantStrength(RGB* d_pixels, int* edgeDir, int* gradiant, int height, int width) {
	int col = blockIdx.x * blockDim.x + threadIdx.x; // width
	int row = blockIdx.y * blockDim.y + threadIdx.y; // height

	if (row >= height || col >= width) {
		return;
	}

	int index = row * width + col;
	float Gx = 0;
	float Gy = 0;
	int newAngle = 2000;

	if (col < width && row < height) {
		if (col > 0 && row > 0 && col < width && row < height) {
			Gx = (-1 * d_pixels[(row - 1) * width + (col - 1)].red) + (-2 * d_pixels[row * width + (col - 1)].red) + (-1 * d_pixels[(row + 1) * width + (col - 1)].red) + (d_pixels[(row - 1) * width + (col + 1)].red) + (2 * d_pixels[row * width + (col + 1)].red) + (d_pixels[(row + 1) * width + (col + 1)].red);
			Gy = (d_pixels[(row - 1) * width + (col - 1)].red) + (2 * d_pixels[(row - 1) * width + col].red) + (d_pixels[(row - 1) * width + (col + 1)].red) + (-1 * d_pixels[(row + 1) * width + (col - 1)].red) + (-2 * d_pixels[(row + 1) * width + col].red) + (-1 * d_pixels[(row + 1) * width + (col + 1)].red);

		}
	}

	gradiant[index] = sqrt((Gx * Gx) + (Gy * Gy));	// Calculate gradient strength						
	double thisAngle = (atan2(Gx, Gy) / 3.14159) * 180.0;		// Calculate actual direction of edge
	//std::cout << "this Angle is: " << thisAngle << " gradiant: " << sqrt((Gx * Gx) + (Gy * Gy)) << std::endl;
	//printf("this angle is: %d, gradient: %d \n", thisAngle, sqrt((Gx * Gx) + (Gy * Gy)));
	/* Convert actual edge direction to approximate value */
	if (((thisAngle < 22.5) && (thisAngle > -22.5)) || (thisAngle > 157.5) || (thisAngle < -157.5)) {
		newAngle = 0;
	}
	if (((thisAngle > 22.5) && (thisAngle < 67.5)) || ((thisAngle < -112.5) && (thisAngle > -157.5))) {
		newAngle = 45;
	}
	if (((thisAngle > 67.5) && (thisAngle < 112.5)) || ((thisAngle < -67.5) && (thisAngle > -112.5))) {
		newAngle = 90;
	}
	if (((thisAngle > 112.5) && (thisAngle < 157.5)) || ((thisAngle < -22.5) && (thisAngle > -67.5))) {
		newAngle = 135;
	}

	edgeDir[index] = newAngle;	// Store the approximate edge direction of each pixel in one array
}


/**
*	Helper function to calculate the number of blocks on an axis based on the total grid size and number of threads in that axis
*/
__host__ int calcBlockDim(int total, int num_threads)
{
	int r = total / num_threads;
	if (total % num_threads != 0) // add one to cover all the threads per block
		++r;
	return r;
}

/**
*	Host function for launching greyscale kernel
*/
__host__ void d_convert_greyscale(RGB* pixel, int height, int width)
{
	RGB* d_pixel;

	hipMalloc(&d_pixel, height * width * sizeof(RGB));
	hipMemcpy(d_pixel, pixel, height * width * sizeof(RGB), hipMemcpyHostToDevice);

	dim3 grid, block;
	block.x = 16;
	block.y = 16;
	grid.x = calcBlockDim(width, block.x);
	grid.y = calcBlockDim(height, block.y);

	greyscaleKernel << <grid, block >> > (d_pixel, height, width);

	hipMemcpy(pixel, d_pixel, height * width * sizeof(RGB), hipMemcpyDeviceToHost);
}

__host__ void d_convert_blur(RGB* pixel, int height, int width)
{
	RGB* d_pixel;

	// allocate and copy memory to the GPU
	hipMalloc(&d_pixel, height * width * sizeof(RGB));
	hipMemcpy(d_pixel, pixel, height * width * sizeof(RGB), hipMemcpyHostToDevice);

	// determine the grid and block for the GPU
	dim3 grid, block;
	block.x = 16;
	block.y = 16;
	grid.x = calcBlockDim(width, block.x);
	grid.y = calcBlockDim(height, block.y);

	// blur the image
	blurKernel << <grid, block >> > (d_pixel, height, width);

	// copy the result back to the CPU
	hipMemcpy(pixel, d_pixel, height * width * sizeof(RGB), hipMemcpyDeviceToHost);
}

__host__ void d_edge_detection(RGB* pixel, int height, int width)
{
	RGB* d_pixel;
	RGB* d_result;

	hipMalloc(&d_pixel, height * width * sizeof(RGB));
	hipMemcpy(d_pixel, pixel, height * width * sizeof(RGB), hipMemcpyHostToDevice);
	hipMalloc(&d_result, height * width * sizeof(RGB));
	hipMemcpy(d_result, pixel, height * width * sizeof(RGB), hipMemcpyHostToDevice);

	dim3 grid, block;
	block.x = 16;
	block.y = 16;
	grid.x = calcBlockDim(width, block.x);
	grid.y = calcBlockDim(height, block.y);

	edgeDetectionKernel << <grid, block >> > (d_pixel, d_result, height, width);

	hipMemcpy(pixel, d_result, height * width * sizeof(RGB), hipMemcpyDeviceToHost);
}

__host__ void d_laplacian(RGB* pixel, int height, int width)
{
	RGB* d_pixel;
	RGB* d_result;

	hipMalloc(&d_pixel, height * width * sizeof(RGB));
	hipMemcpy(d_pixel, pixel, height * width * sizeof(RGB), hipMemcpyHostToDevice);
	hipMalloc(&d_result, height * width * sizeof(RGB));
	hipMemcpy(d_result, pixel, height * width * sizeof(RGB), hipMemcpyHostToDevice);

	dim3 grid, block;
	block.x = 16;
	block.y = 16;
	grid.x = calcBlockDim(width, block.x);
	grid.y = calcBlockDim(height, block.y);

	laplacianKernel << <grid, block >> > (d_pixel, d_result, height, width);

	hipMemcpy(pixel, d_result, height * width * sizeof(RGB), hipMemcpyDeviceToHost);
}

__host__ void d_contrast(RGB* pixel, int height, int width, int rincrease, int gincrease, int bincrease)
{
	RGB* d_pixel;

	hipMalloc(&d_pixel, height * width * sizeof(RGB));
	hipMemcpy(d_pixel, pixel, height * width * sizeof(RGB), hipMemcpyHostToDevice);

	dim3 grid, block;
	block.x = 16;
	block.y = 16;
	grid.x = calcBlockDim(width, block.x);
	grid.y = calcBlockDim(width, block.y);

	contrastKernel << <grid, block >> > (d_pixel, height, width, rincrease, gincrease, bincrease);

	hipMemcpy(pixel, d_pixel, height * width * sizeof(RGB), hipMemcpyDeviceToHost);
}

__host__ void d_brightness(RGB* pixel, int height, int width, int bright)
{
	RGB* d_pixel;

	hipMalloc(&d_pixel, height * width * sizeof(RGB));
	hipMemcpy(d_pixel, pixel, height * width * sizeof(RGB), hipMemcpyHostToDevice);

	dim3 grid, block;
	block.x = 16;
	block.y = 16;
	grid.x = calcBlockDim(width, block.x);
	grid.y = calcBlockDim(width, block.y);

	brightnessKernel << <grid, block >> > (d_pixel, height, width, bright);

	hipMemcpy(pixel, d_pixel, height * width * sizeof(RGB), hipMemcpyDeviceToHost);
}

__host__ void gradiantLauncher(RGB* pixels, int* edgeDir, int* gradiant, int height, int width) {
	RGB* d_pixel;
	int* d_edgeDir;
	int* d_gradiant;

	hipMalloc(&d_pixel, height * width * sizeof(RGB));
	hipMemcpy(d_pixel, pixels, height * width * sizeof(RGB), hipMemcpyHostToDevice);

	hipMalloc(&d_edgeDir, height * width * sizeof(int));
	hipMemcpy(d_edgeDir, edgeDir, height * width * sizeof(int), hipMemcpyHostToDevice);
	hipMalloc(&d_gradiant, height * width * sizeof(int));
	hipMemcpy(d_gradiant, gradiant, height * width * sizeof(int), hipMemcpyHostToDevice);

	dim3 grid, block;
	block.x = 16;
	block.y = 16;
	grid.x = calcBlockDim(width, block.x);
	grid.y = calcBlockDim(height, block.y);

	GradiantStrength << <grid, block >> > (d_pixel, d_edgeDir, d_gradiant, height, width);
	hipMemcpy(pixels, d_pixel, height * width * sizeof(RGB), hipMemcpyDeviceToHost);
	hipMemcpy(edgeDir, d_edgeDir, height * width * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(gradiant, d_gradiant, height * width * sizeof(int), hipMemcpyDeviceToHost);
}